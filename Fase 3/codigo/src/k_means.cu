#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "./k_means.h"

//Host Memory
float *pontoX,*pontoY;
float *centroidX,*centroidY;
int *pontoCluster;

//Device Memory
float *dpontoX, *dpontoY;
float *dcentroidX, *dcentroidY;
int *dpontoCluster;

float *dsumX,*dsumY;
int *dnumPontos;

//Constants and Blocks Size
#define NUM_THREADS_PER_BLOCK 1024
int NUM_BLOCKS_FOR_POINTS,NUM_BLOCKS_FOR_CENTROIDS;
int POINT_ARRAY_SIZE,CENTROID_ARRAY_SIZE,POINTCLUSTER_ARRAY_SIZE;
int N,K;
int nIter = 0;
int nMaxIter = 20;


using namespace std;

void aloca(){
    pontoX = (float *) malloc(N*sizeof(float));
    pontoY = (float *) malloc(N*sizeof(float));
    centroidX = (float *) malloc(K*sizeof(float));
    centroidY = (float *) malloc(K*sizeof(float));
    pontoCluster = (int *) malloc(N*sizeof(int));
}

void inicializa() {
    srand(10);
    for(int i = 0; i < N; i++) {
        pontoX[i] = (float) rand() / RAND_MAX;
        pontoY[i] = (float) rand() / RAND_MAX;
        pontoCluster[i] = -1;
    }
    for(int i = 0; i < K; i++) {
        centroidX[i] = pontoX[i];
        centroidY[i] = pontoY[i];
    }
}


void printFinalResult(){
    //Cálculo do número de pontos por clusters
    int numPontosInCluster[K];
    for(int i = 0; i < K; i++)
        numPontosInCluster[i] = 0;

    for(int i = 0; i < N; i++)
        numPontosInCluster[pontoCluster[i]]++;

    //Imprime toda a informação
    printf("N = %d, K = %d\n",N,K);
    for(int i = 0; i < K; i++)
        printf("Center: (%.3f, %.3f) : Size: %d\n",centroidX[i],centroidY[i],numPontosInCluster[i]);

    printf("Iterations: %d\n",nIter);
}


// -----------------------------------
// -------------  CUDA ---------------
// -----------------------------------
void initializeKernel (){
    // allocate the memory on the device
    hipMalloc ((void**) &dpontoX, POINT_ARRAY_SIZE);
    hipMalloc ((void**) &dpontoY, POINT_ARRAY_SIZE);
    hipMalloc ((void**) &dcentroidX, CENTROID_ARRAY_SIZE);
    hipMalloc ((void**) &dcentroidY, CENTROID_ARRAY_SIZE);
    hipMalloc ((void**) &dpontoCluster, POINTCLUSTER_ARRAY_SIZE);
    hipMalloc ((void**) &dsumX, CENTROID_ARRAY_SIZE);
    hipMalloc ((void**) &dsumY, CENTROID_ARRAY_SIZE);
    hipMalloc ((void**) &dnumPontos,  K * sizeof(int));
    checkCUDAError("mem allocation");

    // Copy all working data to Device
    hipMemcpy (dpontoX,pontoX,POINT_ARRAY_SIZE,hipMemcpyHostToDevice);
    hipMemcpy (dpontoY,pontoY,POINT_ARRAY_SIZE,hipMemcpyHostToDevice);
    hipMemcpy (dcentroidX,centroidX,CENTROID_ARRAY_SIZE,hipMemcpyHostToDevice);
    hipMemcpy (dcentroidY,centroidY,CENTROID_ARRAY_SIZE,hipMemcpyHostToDevice);
    checkCUDAError("memcpy h->d");
}

void freeKernel (){
    // free the device memory
    hipFree(dpontoX); hipFree(dpontoY);
    hipFree(dcentroidX); hipFree(dcentroidY);
    hipFree(dpontoCluster);
    hipFree(dsumX); hipFree(dsumY);
    hipFree(dnumPontos);
    checkCUDAError("mem free");
}

__global__
void atribuiClusterKernel (float *myPontoX, float *myPontoY, int myN, float *myCentroidX, float *myCentroidY, int myK, int *myPontoCluster) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lid = threadIdx.x;
    
    __shared__ float shared_centroidX[NUM_THREADS_PER_BLOCK];
    __shared__ float shared_centroidY[NUM_THREADS_PER_BLOCK];

    if(lid == 0){
        for(int j = 0; j < myK; j++) {
            shared_centroidX[j] = myCentroidX[j];
            shared_centroidY[j] = myCentroidY[j];
        }
    }
    __syncthreads();
    
    if (id >= myN) return;

    float minDistancia = FP_INFINITE;
    int bestCluster = -1;
    //Calcular a distância entre todos os centroides.
    for(int j = 0; j < myK; j++) {
        //Distância euclidiana
        float xSub = (myPontoX[id] - shared_centroidX[j]);
        float ySub = (myPontoY[id] - shared_centroidY[j]);
        float distancia = xSub*xSub + ySub*ySub;
        if (distancia < minDistancia){
            minDistancia = distancia;
            bestCluster = j;
        }
    }

    //Atribuir o melhor cluster ao ponto
    myPontoCluster[id] = bestCluster;
}

__global__
void limpaSumatoriosKernel (int myK, float *mySumX, float *mySumY, int *myNumPontos){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= myK) return;

    mySumX[id] = 0;
    mySumY[id] = 0;
    myNumPontos[id] = 0;
}


__global__
void calculaSomatorioKernel (float *myPontoX, float *myPontoY, int myN,  int *myPontoCluster,
                             float *mySumX, float *mySumY, int *myNumPontos, int myK) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= myN) return;
    
    int clusterID = myPontoCluster[id];
    atomicAdd(&mySumX[clusterID], myPontoX[id]);
    atomicAdd(&mySumY[clusterID], myPontoY[id]);
    atomicAdd(&myNumPontos[clusterID], 1);
}

__global__
void calculaNovosCentroidsKernel (float *myCentroidX, float *myCentroidY, int myK, float *mySumX, float *mySumY, int *myNumPontos){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= myK) return;

    myCentroidX[id] = mySumX[id] / myNumPontos[id];
    myCentroidY[id] = mySumY[id] / myNumPontos[id];
}




int main(int argc,char** argv){
    if (argc != 3){
        printf("Número de argumentos incorretos\n");
        return 1;
    }

    int argNums[2] = {0,0};
    for(int i = 1;i<argc;i++){
        if (sscanf(argv[i],"%d",&argNums[i-1]) != 1){
            printf("Invalid argument '%s'. Only Numbers accepted\n",argv[i]);
            return 1;
        }
    }
    N = argNums[0];
    K = argNums[1];
    if (K > 1024 || N > 1000000 || K <= 0 || N <= 0){
        printf("Invalid argument values.\nValue of N must be between 1 and 1.000.000\nValue of K must be between 1 and 1024\n");
        return 1;
    }

    NUM_BLOCKS_FOR_POINTS = ((int) ceil((double) (N/NUM_THREADS_PER_BLOCK))) + 1;
    NUM_BLOCKS_FOR_CENTROIDS = 1;
    POINT_ARRAY_SIZE = N*sizeof(float);
    CENTROID_ARRAY_SIZE = K*sizeof(float);
    POINTCLUSTER_ARRAY_SIZE = N*sizeof(int);

    aloca();
    inicializa();
    initializeKernel();

    atribuiClusterKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS_FOR_POINTS >>> (dpontoX, dpontoY, N, dcentroidX, dcentroidY, K, dpontoCluster);        
    checkCUDAError("kernel invocation");
    while(nIter < nMaxIter){
        //AJUSTA CENTROIDS
        limpaSumatoriosKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS_FOR_CENTROIDS >>> (K, dsumX, dsumY, dnumPontos);
        calculaSomatorioKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS_FOR_POINTS >>> (dpontoX, dpontoY, N, dpontoCluster, dsumX, dsumY, dnumPontos, K);
        calculaNovosCentroidsKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS_FOR_CENTROIDS >>> (dcentroidX, dcentroidY, K, dsumX, dsumY, dnumPontos);    
        checkCUDAError("kernel invocation");

        atribuiClusterKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS_FOR_POINTS >>> (dpontoX, dpontoY, N, dcentroidX, dcentroidY, K, dpontoCluster);
        checkCUDAError("kernel invocation");        

        nIter++;
    }

    //Copy results from device
    hipMemcpy (centroidX,dcentroidX,CENTROID_ARRAY_SIZE,hipMemcpyDeviceToHost);
    hipMemcpy (centroidY,dcentroidY,CENTROID_ARRAY_SIZE,hipMemcpyDeviceToHost);
    hipMemcpy (pontoCluster,dpontoCluster,POINTCLUSTER_ARRAY_SIZE,hipMemcpyDeviceToHost);
    checkCUDAError("memcpy d->h");

    freeKernel();
    printFinalResult();
}

